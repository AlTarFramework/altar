#include "hip/hip_runtime.h"
// -*- C++ -*-
//
// michael a.g. aïvázis <michael.aivazis@para-sim.com>
// (c) 2013-2020 parasim inc
// all rights reserved
//

// configuration
#include <portinfo>
// cuda
#include <hip/hip_runtime.h>
// external
#include <pyre/journal.h>
#include <gsl/gsl_matrix.h>
// my class declaration
#include "Source.h"


// the displacement kernel
__global__ static
void displacements(
                   // the sizes of things
                   std::size_t nParameters,
                   std::size_t nSamples,
                   std::size_t nObservations,

                   // the elastic constant of the medium
                   double nu,

                   // the current sample set
                   double * theta,
                   // the coordinates of the observation points
                   double * locations,
                   // and the components of the corresponding LOS unit vectors
                   double * los,

                   // the parameter layout within a sample
                   std::size_t xIdx,
                   std::size_t yIdx,
                   std::size_t dIdx,
                   std::size_t sIdx,

                   // the predicted displacements
                   double * predicted
                   );

// the implementation of the source method
void
altar::models::cudamogi::Source::
_displacements() const
{
    // make a channel
    pyre::journal::debug_t channel("cudamogi.source");

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "launching the displacements kernel"
        << pyre::journal::endl;

    // if each block has T threads
    const int T = 128;
    // then we need B blocks to process all the chains
    const int B = _nSamples/T + (_nSamples % T ? 1 : 0);

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "displacements: launching " << B << " blocks of " << T << " threads each"
        << pyre::journal::endl;

    // compute the displacements
    ::displacements<<<B, T>>>(
                          _nParameters, _nSamples, _nObservations,
                          _nu,
                          _samples, _locations, _los,
                          _xIdx, _yIdx, _dIdx, _sIdx,
                          _predicted
                          );

    // wait for the device to finish
    hipError_t status = hipDeviceSynchronize();
    // if something went wrong
    if (status != hipSuccess) {
        // make a channel
        pyre::journal::error_t error("cudamogi.source");
        // complain
        error
            << pyre::journal::at(__HERE__)
            << "while computing the displacements: "
            << hipGetErrorName(status) << " (" << status << ")"
            << pyre::journal::endl;
        // and bail
        throw std::runtime_error("error while computing displacements");
    }

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "displacements kernel done"
        << pyre::journal::endl;

    // all done
    return;
}


// the kernel
__global__ static
void
displacements(
              // the sizes of things
              std::size_t nParameters,
              std::size_t nSamples,
              std::size_t nObservations,

              // the elastic constant of the medium
              double nu,

              // the current sample set
              double * theta,
              // the coordinates of the observation points
              double * locations,
              // and the components of the corresponding LOS unit vectors
              double * los,

              // the parameter layout within a sample
              std::size_t xIdx,
              std::size_t yIdx,
              std::size_t dIdx,
              std::size_t sIdx,

              // the predicted displacements
              double * predicted
              )
{
    // build the workload descriptors
    // global
    // std::size_t B = gridDim.x; // number of blocks
    std::size_t T = blockDim.x;   // number of threads per block
    // std::size_t W = B*T;       // total number of workers
    // local
    std::size_t b = blockIdx.x;   // my block id
    std::size_t t = threadIdx.x;  // my thread id within my block
    std::size_t w = b*T + t;      // my worker id

    // if we have processed all the samples
    if (w >= nSamples) {
        // there is nothing for me to do
        return;
    }

    // i need this
    const auto pi = 4*std::atan(1.0);
    // compute the beginning of my sample
    auto * mine = theta + w*nParameters;

    // get the source location
    auto xSrc = mine[xIdx];
    auto ySrc = mine[yIdx];
    auto dSrc = mine[dIdx];
    // the source strength
    auto sSrc = std::pow(10, mine[sIdx]);

    // go through each observation location
    for (std::size_t loc=0; loc<nObservations; ++loc) {
        // unpack the location of the observation point
        auto xObs = locations[loc];
        auto yObs = locations[nObservations+loc];

        // compute the displacement from the source to the observation point
        auto x = xSrc - xObs;
        auto y = ySrc - yObs;
        auto d = dSrc;

        // compute the distance
        auto R  = std::sqrt(x*x + y*y +d*d);
        // compute the elastic response
        auto C = (nu - 1) * sSrc / pi;
        // form the scaling term
        auto CR3 = C / (R*R*R);

        // compute the components of the unit LOS vector
        auto nx = los[loc];
        auto ny = los[nObservations + loc];
        auto nz = los[2*nObservations + loc];

        // project the displacement to the LOS
        auto u = (x*nx + y*ny - d*nz) * CR3;
        // save
        predicted[loc*nSamples + w] = u;
    }

    // all done
    return;
}


// end of file

#include "hip/hip_runtime.h"
// -*- C++ -*-
//
// michael a.g. aïvázis <michael.aivazis@para-sim.com>
// (c) 2013-2020 parasim inc
// all rights reserved
//

// configuration
#include <portinfo>
// cuda
#include <hip/hip_runtime.h>
// external
#include <pyre/journal.h>
#include <gsl/gsl_matrix.h>
// my class declaration
#include "Source.h"


// the displacement kernel
__global__ static
void residuals(
               // the sizes of things
               std::size_t nParameters,
               std::size_t nSamples,
               std::size_t nObservations,

               // the current sample set
               double * theta,
               // the observed displacements
               double * data,
               // the dataset id map
               std::size_t * oids,

               // the shift parameter index within a sample
               std::size_t offsetIdx,

               // the predicted displacements
               double * predicted
               );

// the implementation of the source method
void
altar::models::cudacdm::Source::
_residuals() const
{
    // make a channel
    pyre::journal::debug_t channel("cudacdm.source");

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "launching the residuals kernel"
        << pyre::journal::endl;

    // if each block has T threads
    const int T = 128;
    // then we need B blocks to process all the chains
    const int B = _nSamples/T + (_nSamples % T ? 1 : 0);

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "residuals: launching " << B << " blocks of " << T << " threads each"
        << pyre::journal::endl;

    // compute the displacements
    ::residuals<<<B, T>>>(
                          _nParameters, _nSamples, _nObservations,
                          _samples, _data, _oids,
                          _offsetIdx,
                          _predicted
                          );

    // wait for the device to finish
    hipError_t status = hipDeviceSynchronize();
    // if something went wrong
    if (status != hipSuccess) {
        // make a channel
        pyre::journal::error_t error("cudacdm.source");
        // complain
        error
            << pyre::journal::at(__HERE__)
            << "while computing the residuals: "
            << hipGetErrorName(status) << " (" << status << ")"
            << pyre::journal::endl;
        // and bail
        throw std::runtime_error("error while computing residuals");
    }

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "residuals kernel done"
        << pyre::journal::endl;

    // all done
    return;
}


// the kernel
__global__ static
void
residuals(
              // the sizes of things
              std::size_t nParameters,
              std::size_t nSamples,
              std::size_t nObservations,

              // the current sample set
              double * theta,
              // the observed displacements
              double * data,
              // the dataset id map
              std::size_t * oids,

              // the shift parameter index within a sample
              std::size_t offsetIdx,

              // the predicted displacements
              double * predicted
              )
{
    // build the workload descriptors
    // global
    // std::size_t B = gridDim.x; // number of blocks
    std::size_t T = blockDim.x;   // number of threads per block
    // std::size_t W = B*T;       // total number of workers
    // local
    std::size_t b = blockIdx.x;   // my block id
    std::size_t t = threadIdx.x;  // my thread id within my block
    std::size_t w = b*T + t;      // my worker id

    // if we have processed all the samples
    if (w >= nSamples) {
        // there is nothing for me to do
        return;
    }

    // compute the beginning of my sample
    auto * mine = theta + w*nParameters;

    // go through the observation points
    for (std::size_t loc=0; loc < nObservations; ++loc) {
        // get the observation
        auto observed = data[loc];
        // lookup  the overall shift that corresponds to this observation
        auto shift = mine[offsetIdx + oids[loc]];
        // adjust the value
        predicted[loc*nSamples + w] -= observed + shift;
    }

    // all done
    return;
}


// end of file

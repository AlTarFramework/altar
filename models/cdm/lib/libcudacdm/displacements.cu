#include "hip/hip_runtime.h"
// -*- C++ -*-
//
// michael a.g. aïvázis <michael.aivazis@para-sim.com>
// (c) 2013-2019 parasim inc
// all rights reserved
//

// configuration
#include <portinfo>
// math
#include <cmath>
#include <limits>
// cuda
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
// external
#include <pyre/journal.h>
#include <gsl/gsl_matrix.h>
// my class declaration
#include "Source.h"

// type aliases
using vec_t = double3;
struct mat_t { double4 rows[3]; };

// the displacement kernel
__global__ static
void displacements(
                   // the sizes of things
                   std::size_t nParameters,
                   std::size_t nSamples,
                   std::size_t nObservations,

                   // the elastic constant of the medium
                   double nu,

                   // the current sample set
                   double * theta,
                   // the coordinates of the observation points
                   double * locations,
                   // and the components of the corresponding LOS unit vectors
                   double * los,

                   // the parameter layout within a sample
                   std::size_t xIdx,
                   std::size_t yIdx,
                   std::size_t dIdx,
                   std::size_t openingIdx,
                   std::size_t aXIdx,
                   std::size_t aYIdx,
                   std::size_t aZIdx,
                   std::size_t omegaXIdx,
                   std::size_t omegaYIdx,
                   std::size_t omegaZIdx,

                   // the predicted displacements
                   double * predicted
                   );

// helpers
__device__ static
void
RDdispSurf(std::size_t w, std::size_t nSamples, std::size_t nObservations,
           const double * locations, double * los,
           const vec_t & P1, const vec_t & P2, const vec_t & P3, const vec_t & P4,
           double opening, double nu,
           double * results);

__device__ static
vec_t AngSetupFSC(double x, double y,
                  const vec_t & b, const vec_t & PA, const vec_t & PB,
                  double nu);

__device__ static
vec_t AngDisDispSurf(const vec_t & y, double beta, const vec_t & b,
               double nu, double a);
// algebra
__device__ inline static vec_t operator+(const vec_t & v);
__device__ inline static vec_t operator-(const vec_t & v);
__device__ inline static vec_t operator+(const vec_t & v1, const vec_t & v2);
__device__ inline static vec_t operator-(const vec_t & v1, const vec_t & v2);
__device__ inline static vec_t operator*(double a, const vec_t & v);
__device__ inline static vec_t operator*(const vec_t & v, double a);
__device__ inline static vec_t operator/(const vec_t & v, double a);

__device__ inline static mat_t operator*(const mat_t & m1, const mat_t & m2);

__device__ inline static double norm(const vec_t & v);
__device__ inline static double dot(const vec_t & v1, const vec_t & v2);
__device__ inline static vec_t cross(const vec_t & v1, const vec_t & v2);
__device__ inline static mat_t transpose(const mat_t & m);
__device__ inline static vec_t xform(const mat_t & m, const vec_t & v);

// trig
__device__ inline static double sind(double);
__device__ inline static double cosd(double);


// the implementation of the source method
void
altar::models::cudacdm::Source::
_displacements() const
{
    // make a channel
    pyre::journal::debug_t channel("cudacdm.source");

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "launching the displacements kernel"
        << pyre::journal::endl;

    // if each block has T threads
    const int T = 64;
    // then we need B blocks to process all the chains
    const int B = _nSamples/T + (_nSamples % T ? 1 : 0);

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "displacements: launching " << B << " blocks of " << T << " threads each"
        << pyre::journal::endl;

    // compute the displacements
    ::displacements<<<B, T>>>(
                          _nParameters, _nSamples, _nObservations,
                          _nu,
                          _samples, _locations, _los,
                          _xIdx, _yIdx, _dIdx, _openingIdx,
                          _aXIdx, _aYIdx, _aZIdx,
                          _omegaXIdx, _omegaYIdx, _omegaZIdx,
                          _predicted
                          );

    // wait for the device to finish
    hipError_t status = hipDeviceSynchronize();
    // if something went wrong
    if (status != hipSuccess) {
        // make a channel
        pyre::journal::error_t error("cudacdm.source");
        // complain
        error
            << pyre::journal::at(__HERE__)
            << "while computing the displacements: "
            << hipGetErrorName(status) << " (" << status << ")"
            << pyre::journal::endl;
        // and bail
        throw std::runtime_error("error while computing displacements");
    }

    // show me
    channel
        << pyre::journal::at(__HERE__)
        << "displacements kernel done"
        << pyre::journal::endl;

    // all done
    return;
}


// the kernel
__global__ static
void
displacements(
              // the sizes of things
              std::size_t nParameters,
              std::size_t nSamples,
              std::size_t nObservations,

              // the elastic constant of the medium
              double nu,

              // the current sample set
              double * theta,
              // the coordinates of the observation points
              double * locations,
              // and the components of the corresponding LOS unit vectors
              double * los,

              // the parameter layout within a sample
              std::size_t xIdx,
              std::size_t yIdx,
              std::size_t dIdx,
              std::size_t openingIdx,
              std::size_t aXIdx,
              std::size_t aYIdx,
              std::size_t aZIdx,
              std::size_t omegaXIdx,
              std::size_t omegaYIdx,
              std::size_t omegaZIdx,

              // the predicted displacements
              double * predicted
              )
{
    // build the workload descriptors
    // global
    // std::size_t B = gridDim.x; // number of blocks
    std::size_t T = blockDim.x;   // number of threads per block
    // std::size_t W = B*T;       // total number of workers
    // local
    std::size_t b = blockIdx.x;   // my block id
    std::size_t t = threadIdx.x;  // my thread id within my block
    std::size_t w = b*T + t;      // my worker id

    // if we have processed all the samples
    if (w >= nSamples) {
        // there is nothing for me to do
        return;
    }

    // compute the beginning of my sample
    auto * mine = theta + w*nParameters;

    // constants
    const double eps = 1e-11;
    // get the source location
    auto x = mine[xIdx];
    auto y = mine[yIdx];
    auto depth = mine[dIdx];
    // the opening
    auto opening = mine[openingIdx];
    // the semi axes
    auto aX = mine[aXIdx];
    auto aY = mine[aXIdx+1];
    auto aZ = mine[aXIdx+2];
    // the orientations
    auto omegaX = mine[omegaXIdx];
    auto omegaY = mine[omegaXIdx + 1];
    auto omegaZ = mine[omegaXIdx + 2];

    mat_t Rx = {make_double4(1.,  0.,           0.,           0.),
                make_double4(0.,  cosd(omegaX), sind(omegaX), 0.),
                make_double4(0., -sind(omegaX), cosd(omegaX), 0.)};

    mat_t Ry = {make_double4(cosd(omegaY), 0., -sind(omegaY), 0.),
                make_double4(0.,           1.,  0.,           0.),
                make_double4(sind(omegaY), 0.,  cosd(omegaY), 0.)};

    mat_t Rz = {make_double4( cosd(omegaZ), sind(omegaZ), 0., 0.),
                make_double4(-sind(omegaZ), cosd(omegaZ), 0., 0.),
                make_double4( 0.,           0.,           1., 0.)};

    // the rotation matrix
    mat_t R = Rz * (Ry * Rx);
    // extract its three columns
    vec_t R_0 = { R.rows[0].x, R.rows[1].x, R.rows[2].x };
    vec_t R_1 = { R.rows[0].y, R.rows[1].y, R.rows[2].y };
    vec_t R_2 = { R.rows[0].z, R.rows[1].z, R.rows[2].z };

    // the centroid
    vec_t P0 = { x, y, -depth };

    vec_t P1 = P0 + (aY*R_1 + aZ*R_2)/2;
    vec_t P2 = P1 - aY*R_1;
    vec_t P3 = P2 - aZ*R_2;
    vec_t P4 = P1 - aZ*R_2;

    vec_t Q1 = P0 + (aZ*R_2 - aX*R_0)/2;
    vec_t Q2 = Q1 + aX*R_0;
    vec_t Q3 = Q2 - aZ*R_2;
    vec_t Q4 = Q1 - aZ*R_2;

    vec_t R1 = P0 + (aX*R_0 + aY*R_1)/2;
    vec_t R2 = R1 - aX*R_0;
    vec_t R3 = R2 - aY*R_1;
    vec_t R4 = R1 - aY*R_1;

    // zero out my slice of the results
    for (auto loc=0; loc<nObservations; ++loc) {
        predicted[loc*nSamples + w] = 0;
    }

    // dispatch the various cases
    if (std::abs(aX) < eps && std::abs(aY) > eps && std::abs(aZ) > eps) {
        RDdispSurf(w, nSamples, nObservations, locations, los,
                   P1, P2, P3, P4, opening, nu, predicted);
    } else if (std::abs(aX) > eps && std::abs(aY) < eps && std::abs(aZ) > eps) {
        RDdispSurf(w, nSamples, nObservations, locations, los,
                   Q1, Q2, Q3, Q4, opening, nu, predicted);
    } else if (std::abs(aX) > eps && std::abs(aY) > eps && std::abs(aZ) < eps) {
        RDdispSurf(w, nSamples, nObservations, locations, los,
                   R1, R2, R3, R4, opening, nu, predicted);
    } else {
    }

    // all done
    return;
}

// helpers
__device__ static
void
RDdispSurf(std::size_t w, std::size_t nSamples, std::size_t nObservations,
           const double * locations, double * los,
           const vec_t & P1, const vec_t & P2, const vec_t & P3, const vec_t & P4,
           double opening, double nu,
           double * results) {
    // opening
    auto V = cross(P2-P1, P4-P1);
    auto b = opening * V/norm(V);

    for (auto loc=0; loc<nObservations; ++loc) {
        // unpack the observation point coordinates
        auto x = locations[loc];
        auto y = locations[nObservations + loc];
        // compute
        auto u1 = AngSetupFSC(x,y, b, P1,P2, nu);
        auto u2 = AngSetupFSC(x,y, b, P2,P3, nu);
        auto u3 = AngSetupFSC(x,y, b, P3,P4, nu);
        auto u4 = AngSetupFSC(x,y, b, P4,P1, nu);

        // assemble
        auto u = u1 + u2 + u3 + u4;
        // compute the unit LOS vector
        vec_t n = { los[loc], los[nObservations + loc], los[2*nObservations + loc] };
        // project the displacement to the LOS
        auto uLOS = dot(u, n);
        // save by accumulating my contribution to the slot
        // N.B.: note the "+=": the general case call this function three times
        results[loc*nSamples + w] += uLOS;
    }

    // all done
    return;
};

__device__ static
vec_t AngSetupFSC(double x, double y,
                  const vec_t & b, const vec_t & PA, const vec_t & PB,
                  double nu) {
    const double pi = HIP_PI;
    vec_t SideVec = PB - PA;
    vec_t eZ = {0, 0, 1};
    auto beta = std::acos(dot(SideVec, eZ) / norm(SideVec));

    if (std::abs(beta) < 1e-12 || std::abs(pi - beta) < 1e-12) {
        return { 0,0,0 };
    }

    vec_t ey1 = { SideVec.x, SideVec.y, 0 };
    ey1 = ey1 / norm(ey1);
    vec_t ey3 = -eZ;
    vec_t ey2 = cross(ey3, ey1);

    mat_t A = { ey1.x, ey1.y, ey1.z,
                ey2.x, ey2.y, ey2.z,
                ey3.x, ey3.y, ey3.z};

    vec_t adcsA = xform(A, {x-PA.x, y-PA.y, -PA.z});
    vec_t adcsAB = xform(A, SideVec);
    vec_t adcsB = adcsA - adcsAB;

    // transform the slip vector
    vec_t bADCS = xform(A, b);

    vec_t vA, vB;
    // distinguish the two configurations
    if (beta*adcsA.x > 0) {
        // configuration I
        vA = AngDisDispSurf(adcsA, -pi+beta, b, nu, -PA.z);
        vB = AngDisDispSurf(adcsB, -pi+beta, b, nu, -PB.z);
    } else {
        // configuration II
        vA = AngDisDispSurf(adcsA, beta, b, nu, -PB.z);
        vB = AngDisDispSurf(adcsB, beta, b, nu, -PB.z);
    }

    vec_t v = xform(transpose(A), vB - vA);

    return v;
}

__device__ static
vec_t AngDisDispSurf(const vec_t & y, double beta, const vec_t & b,
                     double nu, double a)
{
    // constants
    const double pi = HIP_PI;
    // unpack
    auto b1 = b.x;
    auto b2 = b.y;
    auto b3 = b.z;
    auto y1 = y.x;
    auto y2 = y.y;
    // common factors
    auto sinB = std::sin(beta);
    auto cosB = std::cos(beta);
    auto cotB = 1 / std::tan(beta);
    auto z1 = y1*cosB + a*sinB;
    auto z3 = y1*sinB - a*cosB;
    auto r2 = y1*y1 + y2*y2 + a*a;
    auto r = std::sqrt(r2);

    // the Burgers function
    auto Fi = 2*std::atan2(y2, (r+a)/std::tan(beta/2) - y1);

    auto v1b1 = b1/2/pi*((1-(1-2*nu)*cotB*cotB)*Fi +
                         y2/(r+a)*((1-2*nu)*(cotB+y1/2/(r+a))-y1/r) -
                         y2*(r*sinB-y1)*cosB/r/(r-z3));

    auto v2b1 = b1/2/pi*((1-2*nu)*((.5+cotB*cotB)*std::log(r+a)-cotB/sinB*std::log(r-z3)) -
                         1./(r+a)*((1-2*nu)*(y1*cotB-a/2-y2*y2/2/(r+a))+y2*y2/r) +
                         y2*y2*cosB/r/(r-z3));

    auto v3b1 = b1/2/pi*((1-2*nu)*Fi*cotB+y2/(r+a)*(2*nu+a/r) - y2*cosB/(r-z3)*(cosB+a/r));

    auto v1b2 = b2/2/pi*(-(1-2*nu)*((.5-cotB*cotB)*std::log(r+a) + cotB*cotB*cosB*std::log(r-z3) ) -
                         1/(r+a)*((1-2*nu)*(y1*cotB+.5*a+y1*y1/2/(r+a)) - y1*y1/r) +
                         z1*(r*sinB-y1)/r/(r-z3));

    auto v2b2 = b2/2/pi*((1+(1-2*nu)*cotB*cotB)*Fi -
                         y2/(r+a)*((1-2*nu)*(cotB+y1/2/(r+a))-y1/r) -
                         y2*z1/r/(r-z3));

    auto v3b2 = b2/2/pi*(-(1-2*nu)*cotB*(std::log(r+a)-cosB*std::log(r-z3)) -
                         y1/(r+a)*(2*nu+a/r) + z1/(r-z3)*(cosB+a/r));

    auto v1b3 = b3/2/pi*(y2*(r*sinB-y1)*sinB/r/(r-z3));
    auto v2b3 = b3/2/pi*(-y2*y2*sinB/r/(r-z3));
    auto v3b3 = b3/2/pi*(Fi + y2*(r*cosB+a)*sinB/r/(r-z3));

    auto v1 = v1b1 + v1b2 + v1b3;
    auto v2 = v2b1 + v2b2 + v2b3;
    auto v3 = v3b1 + v3b2 + v3b3;

    return {v1, v2, v3};
}



// algebra
__device__ inline static
vec_t operator+(const vec_t & v) {
    return v;
}

__device__ inline static
vec_t operator-(const vec_t & v) {
    return -1.0 *v;
};

__device__ inline static
vec_t operator+(const vec_t & v1, const vec_t & v2) {
    return { v1.x + v2.x, v1.y + v2.y, v1.z + v2.z };
};

__device__ inline static
vec_t operator-(const vec_t & v1, const vec_t & v2) {
    return { v1.x - v2.x, v1.y - v2.y, v1.z - v2.z };
};

__device__ inline static
vec_t operator*(double a, const vec_t & v) {
    return { a*v.x, a*v.y, a*v.z };
};

__device__ inline static
vec_t operator*(const vec_t & v, double a) {
    return { a*v.x, a*v.y, a*v.z };
}

__device__ inline static
vec_t operator/(const vec_t & v, double a) {
    return { v.x/a, v.y/a, v.z/a };
};

__device__ inline static
mat_t operator*(const mat_t & m1, const mat_t & m2) {
    mat_t m;

    m.rows[0] = make_double4(
                             // m[0,0]
                             m1.rows[0].x*m2.rows[0].x +
                             m1.rows[0].y*m2.rows[1].x +
                             m1.rows[0].z*m2.rows[2].x,
                             // m[0,1]
                             m1.rows[0].x*m2.rows[0].y +
                             m1.rows[0].y*m2.rows[1].y +
                             m1.rows[0].z*m2.rows[2].y,
                             // m[0,2]
                             m1.rows[0].x*m2.rows[0].z +
                             m1.rows[0].y*m2.rows[1].z +
                             m1.rows[0].z*m2.rows[2].z,
                             // filler
                             0);

    m.rows[1] = make_double4(
                             // m[1,0]
                             m1.rows[1].x*m2.rows[0].x +
                             m1.rows[1].y*m2.rows[1].x +
                             m1.rows[1].z*m2.rows[2].x,
                             // m[1,1]
                             m1.rows[1].x*m2.rows[0].y +
                             m1.rows[1].y*m2.rows[1].y +
                             m1.rows[1].z*m2.rows[2].y,
                             // m[1,2]
                             m1.rows[1].x*m2.rows[0].z +
                             m1.rows[1].y*m2.rows[1].z +
                             m1.rows[1].z*m2.rows[2].z,
                             // filler
                             0);

    m.rows[2] = make_double4(
                             // m[2,0]
                             m1.rows[2].x*m2.rows[0].x +
                             m1.rows[2].y*m2.rows[1].x +
                             m1.rows[2].z*m2.rows[2].x,
                             // m[2,1]
                             m1.rows[2].x*m2.rows[0].y +
                             m1.rows[2].y*m2.rows[1].y +
                             m1.rows[2].z*m2.rows[2].y,
                             // m[2,2]
                             m1.rows[2].x*m2.rows[0].z +
                             m1.rows[2].y*m2.rows[1].z +
                             m1.rows[2].z*m2.rows[2].z,
                             // filler
                             0);

    return m;
}

__device__ static inline
double norm(const vec_t & v) {
    return std::sqrt(v.x*v.x + v.y*v.y + v.z*v.z);
}

__device__ static inline
double dot(const vec_t & v1, const vec_t & v2) {
    return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__device__ static inline
vec_t cross(const vec_t & v1, const vec_t & v2) {
    return { v1.y*v2.z - v1.z*v2.y, v1.z*v2.x - v1.x*v2.z, v1.x*v2.y - v1.y*v2.x };
}

__device__ static inline
mat_t transpose(const mat_t & m) {
    return {make_double4(m.rows[0].x, m.rows[1].x, m.rows[2].x, 0.),
            make_double4(m.rows[0].y, m.rows[1].y, m.rows[2].y, 0.),
            make_double4(m.rows[0].z, m.rows[1].z, m.rows[2].z, 0.)};
}

__device__ static inline
vec_t xform(const mat_t & m, const vec_t & v) {
    return {m.rows[0].x*v.x + m.rows[0].y*v.y + m.rows[0].z*v.z,
            m.rows[1].x*v.x + m.rows[1].y*v.y + m.rows[1].z*v.z,
            m.rows[2].x*v.x + m.rows[2].y*v.y + m.rows[2].z*v.z};
}

__device__ static inline
double sind(double omega) {
    return std::sin(omega * 180/HIP_PI);
}

__device__ static inline
double cosd(double omega) {
    return std::cos(omega * 180/HIP_PI);
}

// end of file
